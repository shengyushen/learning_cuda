
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<hip/hip_runtime_api.h>


__global__ void VecAdd(float * A, float * B, float * C, int N)
{
	int i= blockDim.x * blockIdx.x+threadIdx.x;
	if(i<N)
		C[i]=A[i]+B[i];
}


int main()
{
	int N=1024;
	size_t size = N*sizeof(float);

	float * h_A=(float*) malloc(size);
	float * h_B=(float*) malloc(size);
	float * h_C=(float*) malloc(size);

	for(int i=0;i<N;i++) {
		h_A[i]=i;
		h_B[N-1-i]=i;
	}

/*	for(int i=0;i<N;i++) {
		printf("i %d h_A[i] %f  h_B[i] %f\n",i,h_A[i],h_B[i]);
	}
*/
	float * d_A;
	hipMalloc(&d_A,size);
	float * d_B;
	hipMalloc(&d_B,size);
	float * d_C;
	hipMalloc(&d_C,size);
	printf("hh1\n");

	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

	printf("hh2\n");
	int threadsPerBlock = 256;
	int blockPerGrid=(N+threadsPerBlock -1)/threadsPerBlock;

	int repeatTime=1024*10;
	hipProfilerStart();
	for(int i=0;i<repeatTime;i++)	{
		VecAdd<<<blockPerGrid,threadsPerBlock>>> (d_A,d_B,d_C,N);
	}
	hipProfilerStop();

	printf("hh3\n");

	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
	printf("hh4\n");
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	printf("hh5\n");

	/*for(int i=0;i<N;i++) {
		printf("i %d h_C[i] %f\n",i,h_C[i]);
	}
	printf("hh6\n");*/

	free(h_A);
	free(h_B);
	free(h_C);
}
