
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<hip/hip_runtime_api.h>


__global__ void VecAdd(float * A, float * B, float * C, int N)
{
	int i= blockDim.x * blockIdx.x+threadIdx.x;
	if(i<N)
		C[i]=A[i]+B[i];
}


int main()
{
	int N=102400;
	size_t size = N*sizeof(float);

	float * xx;

	hipMalloc(&xx,size);

	float * h_A;
	float * h_B;
	float * h_C;

	hipMallocManaged(&h_A,size);
	hipMallocManaged(&h_B,size);
	hipMallocManaged(&h_C,size);

	for(int i=0;i<N;i++) {
		h_A[i]=i;
		h_B[N-1-i]=i;
	}

/*	for(int i=0;i<N;i++) {
		printf("i %d h_A[i] %f  h_B[i] %f\n",i,h_A[i],h_B[i]);
	}
*
	float * d_A;
	cudaMalloc(&d_A,size);
	float * d_B;
	cudaMalloc(&d_B,size);
	float * d_C;
	cudaMalloc(&d_C,size);
	printf("hh1\n");

	cudaMemcpy(d_A,h_A,size,cudaMemcpyHostToDevice);
	cudaMemcpy(d_B,h_B,size,cudaMemcpyHostToDevice);
*/
	float * d_A=h_A;
	float * d_B=h_B;
	float * d_C=h_C;
	printf("hh2\n");
	int threadsPerBlock = 256;
	int blockPerGrid=(N+threadsPerBlock -1)/threadsPerBlock;

	int repeatTime=102400;
//	cudaProfilerStart();
	for(int i=0;i<repeatTime;i++)	{
		VecAdd<<<blockPerGrid,threadsPerBlock>>> (d_A,d_B,d_C,N);
	}
//	cudaProfilerStop();

	printf("hh3\n");

//	cudaMemcpy(h_C,d_C,size,cudaMemcpyDeviceToHost);
	printf("hh4\n");
	
//	cudaFree(d_A);
//	cudaFree(d_B);
//	cudaFree(d_C);
	printf("hh5\n");
	hipDeviceSynchronize();
	for(int i=0;i<N & i< 1024;i++) {
		printf("i %d h_C[i] %f\n",i,h_C[i]);
	}
	printf("hh6\n");

	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);
}
